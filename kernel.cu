#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/driver_types.h"

#include <cstdio>
#include <random>
#include <ctime>
#include <iostream>

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int getRandomValue(const int low, const int high);
void randomizeMatrix(int* matrix, const int size);
void printMatrix(int* matrix, const int size);

// Kernel
__global__ void multiplyKernel(const int* A, const int* B, int* C, const int size);

int main()
{
	constexpr int N = 16;
	constexpr int BYTES = N * N * sizeof(int);
	constexpr int RAND_LOW = 0;
	constexpr int RAND_HIGH = 1;

	srand(time(NULL));

	int* A_host = nullptr;
	int* B_host = nullptr;
	int* C_host = nullptr;

	// Allocate memory on host for A, B, C matrices
	checkCudaErrors(hipHostMalloc(&A_host, BYTES));
	checkCudaErrors(hipHostMalloc(&B_host, BYTES));
	checkCudaErrors(hipHostMalloc(&C_host, BYTES));

	hipDeviceSynchronize();

	randomizeMatrix(A_host, N);
	randomizeMatrix(B_host, N);

	// Zero C-result array XDD
	// memset(C_host, 0, SIZE);

	//printMatrix(A_host, N);
	//printMatrix(B_host, N);
	//printMatrix(C_host, N);

	int* A_device = nullptr;
	int* B_device = nullptr;
	int* C_device = nullptr;

	// Zero copy

	// Allocate memory on device
	checkCudaErrors(hipMalloc(&A_device, BYTES));
	checkCudaErrors(hipMalloc(&B_device, BYTES));
	checkCudaErrors(hipMalloc(&C_device, BYTES));

	// Sync device memory to host memory
	checkCudaErrors(hipHostGetDevicePointer(&A_device, A_host, 0));
	checkCudaErrors(hipHostGetDevicePointer(&B_device, B_host, 0));
	checkCudaErrors(hipHostGetDevicePointer(&C_device, C_host, 0));

	int BLOCK_SIZE = 16;	
	int GRID_SIZE = (int)ceil(N / BLOCK_SIZE);

	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	multiplyKernel <<<grid, threads >>> (A_device, B_device, C_device, N);

	hipDeviceSynchronize();
	printMatrix(C_host, N);

	return 0;
}

__global__ void multiplyKernel(const int* A, const int* B, int* C, const int size)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	C[row * size + col] = 0;
	for (int i = 0; i < size; ++i)
	{
		// printf("A: row=%d,col=%d,i=%d, value=%d\n", row, col, i, A[row * size + i]);
		// printf("B: row=%d,col=%d,i=%d, value=%d\n", row, col, i, B[row * size + i]);
		C[row * size + col] += A[row * size + i] * B[i * size + col];
	}
}

void randomizeMatrix(int* matrix, const int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			matrix[i * size + j] = 1;
		}
	}
}

void printMatrix(int* matrix, const int size)
{
	for (int i = 0; i < size; ++i)
	{
		std::cout << '\n';
		for (int j = 0; j < size; ++j)
		{
			printf("[%d,%d]=%d\n", i, j, matrix[i * size + j]);
		}
	}
	std::cout << '\n';
}

int getRandomValue(const int low, const int high)
{
	return low + static_cast<int>(rand()) * (static_cast<int>(high - low) / RAND_MAX);
}

void verify(int* A, int* B, int* C, const int N)
{
	// For every row...
	for (int i = 0; i < N; i++) {
		// For every column...
		for (int j = 0; j < N; j++) {
			// For every element in the row-column pair
			int tmp = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate the partial results
				tmp += A[i * N + k] * B[k * N + j];
			}

			// Check against the CPU result
			assert(tmp == c[i * N + j]);
		}
	}
}