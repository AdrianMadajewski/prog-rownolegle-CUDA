﻿#include "hip/hip_runtime.h"


#include "hip/driver_types.h"
#include "hip/device_functions.h"

#include <cstdio>
#include <random>
#include <ctime>
#include <iostream>
#include <cassert>

// https://stackoverflow.com/questions/35535831/is-there-any-difference-between-cudamallochost-and-cudahostalloc-without-spe
// https://github.com/CoffeeBeforeArch/cuda_programming/blob/master/02_matrix_mul/tiled/mmul.cu

constexpr int N = 1 << 10;					// 1024
constexpr int SHARED_MEMORY_SIZE = 1 << 10; // 1024

constexpr int BYTES = N * N * sizeof(int);
constexpr int RAND_LOW = 0;
constexpr int RAND_HIGH = 1;

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int getRandomValue(const int low, const int high);
void randomizeMatrix(int* matrix, const int size);
void printMatrix(int* matrix, const int size);
void verify(int* A, int* B, int* C, const int N);

// Kernel
__global__ void multiplyKernel(const int* __restrict__ A, 
	const int* __restrict__ B, 
	int* __restrict__ C, 
	const int size);

int main(int argc, char **argv)
{
	srand(time(NULL));

	int* A_host = nullptr;
	int* B_host = nullptr;
	int* C_host = nullptr;

	// Allocate memory on host for A, B, C matrices
	// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#shared-memory-and-memory-banks

	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
	checkCudaErrors(hipHostAlloc(&A_host, BYTES, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&B_host, BYTES, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&C_host, BYTES, hipHostMallocMapped));

	// cudaDeviceSynchronize();

	randomizeMatrix(A_host, N);
	randomizeMatrix(B_host, N);

	int* A_device = nullptr;
	int* B_device = nullptr;
	int* C_device = nullptr;

	// Zero copy

	// Allocate memory on device
	checkCudaErrors(hipMalloc(&A_device, BYTES));
	checkCudaErrors(hipMalloc(&B_device, BYTES));
	checkCudaErrors(hipMalloc(&C_device, BYTES));

	// Sync device memory to host memory
	checkCudaErrors(hipHostGetDevicePointer(&A_device, A_host, 0));
	checkCudaErrors(hipHostGetDevicePointer(&B_device, B_host, 0));
	checkCudaErrors(hipHostGetDevicePointer(&C_device, C_host, 0));

	int BLOCK_SIZE = 64;	
	int GRID_SIZE = (int)ceil(N / BLOCK_SIZE);

	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	multiplyKernel <<<grid, threads >>> (A_device, B_device, C_device, N);

	hipDeviceSynchronize();

	verify(A_host, B_host, C_host, N);
	// printMatrix(C_host, N);

	printf("COMPLETED WITH SUCCESS\n");
	return 0;
}

__global__ void multiplyKernel(const int* __restrict__ A, 
	const int* __restrict__ B, 
	int* __restrict__ C, 
	const int size)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int shared_A[SHARED_MEMORY_SIZE];
	__shared__ int shared_B[SHARED_MEMORY_SIZE];

	// No alliasing
	int sum = 0;
	for (int i = 0; i < size; i+= blockDim.x)
	{
		// Load elements from tile to shared memory
		int access_tile = threadIdx.y * blockDim.x + threadIdx.x;
		shared_A[access_tile] = A[row * N + i + threadIdx.x];
		shared_B[access_tile] = B[i * N + threadIdx.y * N + col];

		// Intellisense treats it as undefined :(
		__syncthreads();

		// Matrix multiply
		for (int j = 0; j < blockDim.x; j++)
		{
			sum += shared_A[threadIdx.y * blockDim.x + j] * shared_B[j * blockDim.x + threadIdx.x];
		}

		// Intellisense treats it as undefined :(
		__syncthreads();
	}

	// Assign computed result to matrix
	C[row * size + col] = sum;
}

void randomizeMatrix(int* matrix, const int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			matrix[i * size + j] = 1;
		}
	}
}

void printMatrix(int* matrix, const int size)
{
	for (int i = 0; i < size; ++i)
	{
		std::cout << '\n';
		for (int j = 0; j < size; ++j)
		{
			printf("[%d,%d]=%d\n", i, j, matrix[i * size + j]);
		}
	}
	std::cout << '\n';
}

int getRandomValue(const int low, const int high)
{
	return low + static_cast<int>(rand()) * 
		(static_cast<int>(high - low) / RAND_MAX);
}

void verify(int* A, int* B, int* C, const int N)
{
	// For every row...
	for (int i = 0; i < N; i++) {
		// For every column...
		for (int j = 0; j < N; j++) {
			// For every element in the row-column pair
			int tmp = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate the partial results
				tmp += A[i * N + k] * B[k * N + j];
			}

			// Check against the CPU result
			assert(tmp == C[i * N + j]);
		}
	}
}